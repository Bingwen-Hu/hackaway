#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void gpuAdd(int d_a, int d_b, int *d_c)
{
    *d_c = d_a + d_b;
}

int main(void){
    int h_c; // define host variable to store answer
    int *d_c; // define device pointer
    hipMalloc((void**)&d_c, sizeof(int));
    // kernel call by passing 1 and 4 as inputs and storing answer in d_c
    // << <1,1> >> means 1 block is executed with 1 thread per block
    gpuAdd <<<1,1 >>> (1, 4, d_c);
    // copy result from device memory to host memory
    hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);
    printf("1 + 4 = %d\n", h_c);
    hipFree(d_c);
    return 0;
}
