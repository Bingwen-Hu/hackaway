#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void gpuAdd(int *d_a, int *d_b, int *d_c)
{
    *d_c = *d_a + *d_b;
}


int main(void){
    // define
    int h_a, h_b, h_c;
    int *d_a, *d_b, *d_c;

    // init
    h_a = 1;
    h_b = 4;
    
    // alloc
    hipMalloc((void**)&d_a, sizeof(int));
    hipMalloc((void**)&d_b, sizeof(int));
    hipMalloc((void**)&d_c, sizeof(int));

    // copy value of host variable in device memory
    hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice);

    gpuAdd << <1, 1>> > (d_a, d_b, d_c);

    hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);
    printf("Passing Parameter by Reference Output: %d + %d = %d\n", 
            h_a, h_b, h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}
