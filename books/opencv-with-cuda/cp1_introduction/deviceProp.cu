#include <hip/hip_runtime.h>
#include <stdio.h>


int main(){
    int device = 0;
    hipDeviceProp_t device_property;
    hipGetDeviceProperties(&device_property, device);
    printf("\nDevice %d: %s", device, device_property.name);

    int driver_version;
    int runtime_version;
    hipDriverGetVersion(&driver_version);
    hipRuntimeGetVersion(&runtime_version);
    printf("\nCUDA driver Version / Runtime Version %i / %i", 
            driver_version, runtime_version);
    
    return 0;
}
